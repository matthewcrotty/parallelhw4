
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" double* input_data;
extern "C" double* output_data;
extern "C" double my_sum;


struct SharedMemory {
    __device__ inline operator double *() {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};


__device__ __forceinline__ double warpReduceSum(unsigned int mask, double mySum) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        mySum += __shfl_down_sync(mask, mySum, offset);
    }
    return mySum;
}


template <unsigned int blockSize>
__global__ void reduce7(const double *__restrict__ g_idata, double *__restrict__ g_odata,
            unsigned int n) {
    double *sdata = SharedMemory();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int gridSize = blockSize * gridDim.x;
    unsigned int maskLength = (blockSize & 31);  // 31 = warpSize-1
    maskLength = (maskLength > 0) ? (32 - maskLength) : maskLength;
    const unsigned int mask = (0xffffffff) >> maskLength;

    double mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread

    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
        mySum += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for
        // powerOf2 sized arrays
        if ((i + blockSize) < n) {
            mySum += g_idata[i + blockSize];
        }
        i += gridSize;
    }

    // Reduce within warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
    // SM 8.0
    mySum = warpReduceSum(mask, mySum);

    // each thread puts its local sum into shared memory
    if ((tid % warpSize) == 0) {
        sdata[tid / warpSize] = mySum;
    }

    __syncthreads();

    const unsigned int shmem_extent = (blockSize / warpSize) > 0 ? (blockSize / warpSize) : 1;
    const unsigned int ballot_result = __ballot_sync(mask, tid < shmem_extent);
    if (tid < shmem_extent) {
        mySum = sdata[tid];
        // Reduce final warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
        // SM 8.0
        mySum = warpReduceSum(ballot_result, mySum);
    }

    // write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = mySum;
    }
}


extern "C" void initCuda(int my_rank, int num_elements){
    int cudaDeviceCount = 0;
    hipError_t cE;
    if( (cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess){
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount);
        exit(-1);
    }

    if( (cE = hipSetDevice(my_rank % cudaDeviceCount)) != hipSuccess){
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", my_rank, (my_rank % cudaDeviceCount), cE);
        exit(-1);
    }

    printf("Mapping Rank %d to CUDA Device %d \n", my_rank, (my_rank % cudaDeviceCount));

    hipMallocManaged(&input_data, num_elements * sizeof(double));
    hipMallocManaged(&output_data, num_elements * sizeof(double));

    for(int i = 0; i < num_elements; i++){
        input_data[i] = i + (num_elements * my_rank);
        output_data[i] = 0.0;
    }

}

extern "C" void reduceCuda(int num_elements, int threads, int blocks, double* input, double* output){
    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);
    int smemSize = ((threads / 32) + 1) * sizeof(double);

    reduce7<512><<<dimGrid, dimBlock, smemSize>>>(input_data, output_data, num_elements);

    for(int i = 0; i < num_elements/512; i++){
        my_sum += output_data[i];
    }
    printf("Cuda sum %f\n", my_sum);


    hipFree(input);

}
